#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>

__global__ void set_distance(int *dist, int vertices, int src)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i == src)
	{
		dist[i] = 0;
	}
	else if (i < vertices)
	{
		dist[i] = -1;
	}
}

__global__ void update_distances(int *dist, int *C, int *R, int vertices, int iteration, int *done)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x, j;
	if (i < vertices)
	{
		if (dist[i] == iteration)
		{
			*done = 0;
			for (int offset = R[i]; offset < R[i + 1]; ++offset)
			{
				j = C[offset];
				if (dist[j] == -1)
				{
					dist[j] = iteration + 1;
				}
			}
		}
	}
}

/// returns array of distances from source
int* quadratic_parallel_BFS(int *h_C, int *h_R, int edges, int vertices, int src)
{
	// allocate CUDA memory
	int *d_dist, *d_C, *d_R;

	hipMalloc(&d_dist, sizeof(int) * vertices);
	hipMalloc(&d_C, sizeof(int) * edges);
	hipMalloc(&d_R, sizeof(int) * vertices);

	// copy data to device memory
	hipMemcpy(d_C, h_C, sizeof(int) * edges, hipMemcpyHostToDevice);
	hipMemcpy(d_R, h_R, sizeof(int) * vertices, hipMemcpyHostToDevice);

	// set blocks $ threads number
	dim3 threads_per_block(1024);
	dim3 num_blocks(1 + vertices / threads_per_block.x);

	// set initial distances
	set_distance << <num_blocks, threads_per_block >> >(d_dist, vertices, src);

	// update distances
	int iteration = 0, h_done, *d_done_ptr;
	hipMalloc(&d_done_ptr, sizeof(int));

	do
	{
		hipMemset(d_done_ptr, 1, sizeof(int));
		hipMemcpy(&h_done, d_done_ptr, sizeof(int), hipMemcpyDeviceToHost);

		update_distances << <num_blocks, threads_per_block >> >(d_dist, d_C, d_R, vertices, iteration, d_done_ptr);
		hipMemcpy(&h_done, d_done_ptr, sizeof(int), hipMemcpyDeviceToHost);
		++iteration;
	} while (h_done == 0);

	// copy result from device to host
	int *h_dist = (int*)malloc(sizeof(int) * vertices);
	hipMemcpy(h_dist, d_dist, sizeof(int) * vertices, hipMemcpyDeviceToHost);
	return h_dist;
}